#include "hip/hip_runtime.h"
#include "pbrt/films/rgb_film.h"

#include "pbrt/scene/parameter_dictionary.h"
#include "pbrt/spectrum_util/global_spectra.h"

static __global__ void init_pixels(Pixel *pixels, Point2i dimension) {
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= dimension.x * dimension.y) {
        return;
    }

    pixels[idx].init_zero();
}

RGBFilm *RGBFilm::create(const ParameterDictionary &parameters,
                         std::vector<void *> &gpu_dynamic_pointers) {
    auto resolution_x = parameters.get_integer("xresolution")[0];
    auto resolution_y = parameters.get_integer("yresolution")[0];

    auto film_resolution = Point2i(resolution_x, resolution_y);

    FloatType iso = 100;
    FloatType white_balance_val = 0.0;
    FloatType exposure_time = 1.0;
    FloatType imaging_ratio = exposure_time * iso / 100.0;

    auto d_illum =
        Spectrum::create_cie_d(white_balance_val == 0.0 ? 6500.0 : white_balance_val, CIE_S0,
                               CIE_S1, CIE_S2, CIE_S_lambda, gpu_dynamic_pointers);

    PixelSensor *sensor;
    CHECK_CUDA_ERROR(hipMallocManaged(&sensor, sizeof(PixelSensor)));
    gpu_dynamic_pointers.push_back(sensor);

    sensor->init_cie_1931(parameters.global_spectra->cie_xyz,
                          parameters.global_spectra->rgb_color_space,
                          white_balance_val == 0 ? nullptr : d_illum, imaging_ratio);

    Pixel *gpu_pixels;
    CHECK_CUDA_ERROR(
        hipMallocManaged(&gpu_pixels, sizeof(Pixel) * film_resolution.x * film_resolution.y));
    gpu_dynamic_pointers.push_back(gpu_pixels);

    {
        uint threads = 1024;
        uint blocks = divide_and_ceil(uint(film_resolution.x * film_resolution.y), threads);

        init_pixels<<<blocks, threads>>>(gpu_pixels, film_resolution);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    }

    RGBFilm *rgb_film;
    CHECK_CUDA_ERROR(hipMallocManaged(&rgb_film, sizeof(RGBFilm)));
    gpu_dynamic_pointers.push_back(rgb_film);

    rgb_film->init(gpu_pixels, sensor, film_resolution, parameters.global_spectra->rgb_color_space);

    return rgb_film;
}
