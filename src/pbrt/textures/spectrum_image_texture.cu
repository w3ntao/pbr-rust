#include "pbrt/textures/spectrum_image_texture.h"
#include "pbrt/spectra/rgb_albedo_spectrum.h"

const SpectrumImageTexture *SpectrumImageTexture::create(const ParameterDictionary &parameters,
                                                         std::vector<void *> &gpu_dynamic_pointers,
                                                         const RGBColorSpace *_color_space) {
    SpectrumImageTexture *texture;
    CHECK_CUDA_ERROR(hipMallocManaged(&texture, sizeof(SpectrumImageTexture)));
    texture->init(parameters, gpu_dynamic_pointers, _color_space);

    gpu_dynamic_pointers.push_back(texture);

    return texture;
}

void SpectrumImageTexture::init(const ParameterDictionary &parameters,
                                std::vector<void *> &gpu_dynamic_pointers,
                                const RGBColorSpace *_color_space) {
    mipmap = MIPMap::create(parameters, gpu_dynamic_pointers, _color_space);

    color_space = _color_space;
    spectrum_type = SpectrumType::Albedo;

    mapping = UVMapping(parameters);

    scale = parameters.get_float("scale", 1.0);
    invert = parameters.get_bool("invert", false);
}

PBRT_CPU_GPU
SampledSpectrum SpectrumImageTexture::evaluate(const TextureEvalContext &ctx,
                                               const SampledWavelengths &lambda) const {
    auto c = mapping.map(ctx);
    c.st[1] = 1.0 - c.st[1];

    auto _rgb = scale * mipmap->filter(c.st);
    auto rgb = (invert ? RGB(1.0, 1.0, 1.0) - _rgb : _rgb).clamp(0.0, Infinity);

    switch (spectrum_type) {
    case (SpectrumType::Albedo): {
        RGBAlbedoSpectrum rgb_albedo_spectrum;
        rgb_albedo_spectrum.init(rgb.clamp(0.0, 1.0), color_space);
        return rgb_albedo_spectrum.sample(lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}
