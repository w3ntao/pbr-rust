#include "pbrt/lights/image_infinite_light.h"

#include "pbrt/euclidean_space/bounds3.h"
#include "pbrt/euclidean_space/vector3.h"

#include "pbrt/gpu/global_variable.h"

#include "pbrt/scene/parameter_dictionary.h"

#include "pbrt/spectrum_util/rgb_color_space.h"
#include "pbrt/spectrum_util/sampled_spectrum.h"

#include "pbrt/textures/gpu_image.h"
#include "pbrt/util/macro.h"
#include "pbrt/util/math.h"
#include "pbrt/spectra/rgb_illuminant_spectrum.h"

ImageInfiniteLight *ImageInfiniteLight::create(const Transform &_render_from_light,
                                               const ParameterDictionary &parameters,
                                               std::vector<void *> &gpu_dynamic_pointers) {

    auto texture_file = parameters.root + "/" + parameters.get_string("filename", std::nullopt);

    auto scale = parameters.get_float("scale", 1.0);

    const Spectrum *cie_xyz[3];
    parameters.global_variables->get_cie_xyz(cie_xyz);
    const auto cie_y = cie_xyz[1];

    scale /= parameters.global_variables->rgb_color_space->illuminant->to_photometric(cie_y);

    ImageInfiniteLight *image_infinite_light;
    CHECK_CUDA_ERROR(hipMallocManaged(&image_infinite_light, sizeof(ImageInfiniteLight)));
    gpu_dynamic_pointers.push_back(image_infinite_light);

    image_infinite_light->light_type = LightType::infinite;
    image_infinite_light->render_from_light = _render_from_light;

    image_infinite_light->image = GPUImage::create_from_file(texture_file, gpu_dynamic_pointers);
    image_infinite_light->scale = scale;

    image_infinite_light->color_space = parameters.global_variables->rgb_color_space;

    image_infinite_light->scene_radius = NAN;
    image_infinite_light->scene_center = Point3f(NAN, NAN, NAN);

    return image_infinite_light;
}

PBRT_GPU
SampledSpectrum ImageInfiniteLight::le(const Ray &ray, const SampledWavelengths &lambda) const {
    Vector3f wLight = (render_from_light.apply_inverse(ray.d)).normalize();
    auto uv = EqualAreaSphereToSquare(wLight);
    return ImageLe(uv, lambda);
}

PBRT_GPU
cuda::std::optional<LightLiSample> ImageInfiniteLight::sample_li(const LightSampleContext &ctx,
                                                                 const Point2f &u,
                                                                 SampledWavelengths &lambda) const {
    // Convert infinite light sample point to direction
    Vector3f wLight = EqualAreaSquareToSphere(u);
    Vector3f wi = render_from_light(wLight);

    // Compute PDF for sampled infinite light direction
    FloatType pdf = 1.0 / (4 * compute_pi());

    const auto interaction = Interaction(ctx.p() + wi * (2 * scene_radius));

    // Return radiance value for infinite light direction
    return LightLiSample(ImageLe(u, lambda), wi, pdf, interaction);
}

void ImageInfiniteLight::preprocess(const Bounds3f &scene_bounds) {
    scene_bounds.bounding_sphere(&scene_center, &scene_radius);
}

PBRT_GPU
SampledSpectrum ImageInfiniteLight::ImageLe(Point2f uv, const SampledWavelengths &lambda) const {
    auto rgb = image->bilerp(uv, WrapMode::OctahedralSphere);

    // rgb = rgb.clamp(0, Infinity);
    // TODO: without clamp(), the result looks closer to original PBRT

    auto spec = RGBIlluminantSpectrum(rgb, color_space);

    return scale * spec.sample(lambda);
}
