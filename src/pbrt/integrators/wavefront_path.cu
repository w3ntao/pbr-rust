#include "hip/hip_runtime.h"
#include "pbrt/accelerator/hlbvh.h"
#include "pbrt/base/film.h"
#include "pbrt/base/integrator_base.h"
#include "pbrt/base/light.h"
#include "pbrt/base/material.h"
#include "pbrt/base/sampler.h"
#include "pbrt/gui/gl_object.h"
#include "pbrt/integrators/wavefront_path.h"
#include "pbrt/light_samplers/power_light_sampler.h"
#include "pbrt/samplers/independent.h"
#include "pbrt/samplers/stratified.h"
#include "pbrt/scene/parameter_dictionary.h"
#include "pbrt/spectrum_util/sampled_spectrum.h"
#include "pbrt/spectrum_util/sampled_wavelengths.h"
#include "pbrt/util/basic_math.h"

const uint PATH_POOL_SIZE = 2 * 1024 * 1024;

struct FrameBuffer {
    uint pixel_idx;
    uint sample_idx;
    SampledSpectrum radiance;
    SampledWavelengths lambda;
    FloatType weight;
};

struct FBComparator {
    bool operator()(FrameBuffer const &left, FrameBuffer const &right) const {
        if (left.pixel_idx < right.pixel_idx) {
            return true;
        }

        if (left.pixel_idx > right.pixel_idx) {
            return false;
        }

        return left.sample_idx < right.sample_idx;
    }
};

struct MISParameter {
    bool specular_bounce = true;
    bool any_non_specular_bounces = false;

    FloatType pdf_bsdf;
    FloatType eta_scale;
    LightSampleContext prev_interaction_light_sample_ctx;

    PBRT_CPU_GPU
    void init() {
        specular_bounce = true;
        any_non_specular_bounces = false;

        pdf_bsdf = NAN;
        eta_scale = 1.0;
    }
};

static __global__ void gpu_init_independent_samplers(Sampler *samplers,
                                                     IndependentSampler *independent_samplers,
                                                     uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    samplers[worker_idx].init(&independent_samplers[worker_idx]);
}

static __global__ void gpu_init_stratified_samplers(Sampler *samplers,
                                                    StratifiedSampler *stratified_samplers,
                                                    uint samples_per_dimension, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    stratified_samplers[worker_idx].init(samples_per_dimension);

    samplers[worker_idx].init(&stratified_samplers[worker_idx]);
}

static __global__ void gpu_init_path_state(PathState *path_state) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }

    path_state->init_new_path(worker_idx);
}

__global__ void control_logic(const WavefrontPathIntegrator *integrator, PathState *path_state,
                              Queues *queues) {
    const uint path_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (path_idx >= PATH_POOL_SIZE || path_state->finished[path_idx]) {
        return;
    }

    // otherwise beta is larger than 0.0
    auto &isect = path_state->shape_intersections[path_idx].interaction;
    const auto ray = path_state->camera_rays[path_idx].ray;
    auto &lambda = path_state->lambdas[path_idx];

    const auto path_length = path_state->path_length[path_idx];
    const auto specular_bounce = path_state->mis_parameters[path_idx].specular_bounce;
    auto &beta = path_state->beta[path_idx];
    auto &L = path_state->L[path_idx];

    const auto intersected = path_state->intersected[path_idx];

    const auto prev_interaction_light_sample_ctx =
        path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx;
    const auto pdf_bsdf = path_state->mis_parameters[path_idx].pdf_bsdf;

    bool should_terminate_path =
        !intersected || path_length > integrator->max_depth || !beta.is_positive();

    if (!should_terminate_path && path_length > 8) {
        // possibly terminate the path with Russian roulette

        auto &eta_scale = path_state->mis_parameters[path_idx].eta_scale;
        auto &sampler = path_state->samplers[path_idx];
        const auto u = sampler.get_1d();
        // consume this random value anyway to keep samples aligned

        SampledSpectrum russian_roulette_beta = beta * eta_scale;
        if (russian_roulette_beta.max_component_value() < 1) {
            auto q = clamp<FloatType>(1 - russian_roulette_beta.max_component_value(), 0, 0.95);
            if (u < q) {
                beta = SampledSpectrum(0.0);
                should_terminate_path = true;
            } else {
                beta /= 1 - q;
            }
        }
    }

    if (should_terminate_path) {
        if (beta.is_positive()) {
            // sample infinite lights
            for (uint idx = 0; idx < integrator->base->infinite_light_num; ++idx) {
                auto light = integrator->base->infinite_lights[idx];
                auto Le = light->le(ray, lambda);

                if (path_length == 0 || specular_bounce) {
                    L += beta * Le;
                } else {
                    // Compute MIS weight for infinite light
                    FloatType pdf_light =
                        integrator->base->light_sampler->pmf(prev_interaction_light_sample_ctx,
                                                             light) *
                        light->pdf_li(prev_interaction_light_sample_ctx, ray.d, true);
                    FloatType weight_bsdf = power_heuristic(1, pdf_bsdf, 1, pdf_light);

                    L += beta * weight_bsdf * Le;
                }
            }
        }

        const uint queue_idx = atomicAdd(&queues->frame_buffer_counter, 1);
        queues->frame_buffer_queue[queue_idx] = FrameBuffer{
            .pixel_idx = path_state->pixel_indices[path_idx],
            .sample_idx = path_state->sample_indices[path_idx],
            .radiance = L * path_state->camera_rays[path_idx].weight,
            .lambda = lambda,
            .weight = path_state->camera_samples[path_idx].filter_weight,
        };

        queues->new_path_queue[atomicAdd(&queues->new_path_counter, 1)] = path_idx;
        return;
    }

    SampledSpectrum Le = isect.le(-ray.d, lambda);
    if (Le.is_positive()) {
        if (path_length == 0 || specular_bounce)
            path_state->L[path_idx] += beta * Le;
        else {
            // Compute MIS weight for area light
            auto area_light = isect.area_light;

            FloatType pdf_light = integrator->base->light_sampler->pmf(
                                      prev_interaction_light_sample_ctx, area_light) *
                                  area_light->pdf_li(prev_interaction_light_sample_ctx, ray.d);
            FloatType weight_light = power_heuristic(1, pdf_bsdf, 1, pdf_light);

            path_state->L[path_idx] += beta * weight_light * Le;
        }
    }

    // for active paths: advance one segment

    path_state->path_length[path_idx] += 1;

    switch (isect.material->get_material_type()) {

    case Material::Type::conductor: {
        const uint queue_idx = atomicAdd(&queues->conductor_material_counter, 1);
        queues->conductor_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::coated_conductor: {
        const uint queue_idx = atomicAdd(&queues->coated_conductor_material_counter, 1);
        queues->coated_conductor_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::coated_diffuse: {
        const uint queue_idx = atomicAdd(&queues->coated_diffuse_material_counter, 1);
        queues->coated_diffuse_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::dielectric: {
        const uint queue_idx = atomicAdd(&queues->dielectric_material_counter, 1);
        queues->dielectric_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::diffuse: {
        const uint queue_idx = atomicAdd(&queues->diffuse_material_counter, 1);
        queues->diffuse_material_queue[queue_idx] = path_idx;
        break;
    }

    case Material::Type::mix: {
        printf("\nyou should not see MixMaterial here\n\n");
        REPORT_FATAL_ERROR();
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }
}

__global__ void write_frame_buffer(Film *film, Queues *queues) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->frame_buffer_counter) {
        return;
    }

    const auto pixel_idx = queues->frame_buffer_queue[queue_idx].pixel_idx;
    if (queue_idx > 0 && pixel_idx == queues->frame_buffer_queue[queue_idx - 1].pixel_idx) {
        return;
    }

    for (uint idx = queue_idx; idx < queues->frame_buffer_counter &&
                               queues->frame_buffer_queue[idx].pixel_idx == pixel_idx;
         ++idx) {
        // make sure the same pixels are written by the same thread
        const auto &frame_buffer = queues->frame_buffer_queue[idx];
        film->add_sample(frame_buffer.pixel_idx, frame_buffer.radiance, frame_buffer.lambda,
                         frame_buffer.weight);
    }
}

__global__ void fill_new_path_queue(Queues *queues) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= PATH_POOL_SIZE) {
        return;
    }
    queues->new_path_queue[worker_idx] = worker_idx;
}

__global__ void generate_new_path(const IntegratorBase *base, PathState *path_state,
                                  Queues *queues) {
    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= queues->new_path_counter) {
        return;
    }

    const uint path_idx = queues->new_path_queue[queue_idx];

    const auto unique_path_id = atomicAdd(&path_state->global_path_counter, 1);
    if (unique_path_id >= path_state->total_path_num) {
        path_state->finished[path_idx] = true;
        return;
    }

    const uint width = path_state->image_resolution.x;
    const uint height = path_state->image_resolution.y;

    const uint pixel_idx = unique_path_id % (width * height);
    const uint sample_idx = unique_path_id / (width * height);

    auto sampler = &path_state->samplers[path_idx];

    auto p_pixel = Point2i(pixel_idx % width, pixel_idx / width);

    sampler->start_pixel_sample(pixel_idx, sample_idx, 0);

    path_state->camera_samples[path_idx] = sampler->get_camera_sample(p_pixel, base->filter);
    auto lu = sampler->get_1d();
    path_state->lambdas[path_idx] = SampledWavelengths::sample_visible(lu);

    path_state->camera_rays[path_idx] =
        base->camera->generate_ray(path_state->camera_samples[path_idx], sampler);

    path_state->pixel_indices[path_idx] = pixel_idx;
    path_state->sample_indices[path_idx] = sample_idx;
    path_state->path_length[path_idx] = 0;

    path_state->init_new_path(path_idx);

    uint ray_queue_idx = atomicAdd(&queues->ray_counter, 1);
    queues->ray_queue[ray_queue_idx] = path_idx;
}

template <Material::Type material_type>
__global__ void gpu_evaluate_material(const WavefrontPathIntegrator *integrator,
                                      PathState *path_state, Queues *queues) {
    uint material_counter = 0;
    uint *material_queue = nullptr;

    switch (material_type) {
    case Material::Type::coated_conductor: {
        material_counter = queues->coated_conductor_material_counter;
        material_queue = queues->coated_conductor_material_queue;
        break;
    }

    case Material::Type::coated_diffuse: {
        material_counter = queues->coated_diffuse_material_counter;
        material_queue = queues->coated_diffuse_material_queue;
        break;
    }

    case Material::Type::conductor: {
        material_counter = queues->conductor_material_counter;
        material_queue = queues->conductor_material_queue;
        break;
    }

    case Material::Type::dielectric: {
        material_counter = queues->dielectric_material_counter;
        material_queue = queues->dielectric_material_queue;
        break;
    }

    case Material::Type::diffuse: {
        material_counter = queues->diffuse_material_counter;
        material_queue = queues->diffuse_material_queue;
        break;
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }

    const uint queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (queue_idx >= material_counter) {
        return;
    }

    const uint path_idx = material_queue[queue_idx];

    auto &lambda = path_state->lambdas[path_idx];

    auto sampler = &path_state->samplers[path_idx];

    auto &isect = path_state->shape_intersections[path_idx].interaction;

    path_state->bsdf[path_idx] =
        isect.get_bsdf(lambda, integrator->base->camera, sampler->get_samples_per_pixel());

    integrator->sample_bsdf(path_idx, path_state);

    uint ray_queue_idx = atomicAdd(&queues->ray_counter, 1);
    queues->ray_queue[ray_queue_idx] = path_idx;
}

__global__ void ray_cast(const WavefrontPathIntegrator *integrator, PathState *path_state,
                         Queues *queues) {
    const uint ray_queue_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ray_queue_idx >= queues->ray_counter) {
        return;
    }

    const uint path_idx = queues->ray_queue[ray_queue_idx];

    const auto camera_ray = path_state->camera_rays[path_idx];

    auto intersection = integrator->base->intersect(camera_ray.ray, Infinity);

    path_state->intersected[path_idx] = intersection.has_value();

    if (intersection.has_value()) {
        path_state->shape_intersections[path_idx] = intersection.value();
    }
}

PBRT_GPU void WavefrontPathIntegrator::sample_bsdf(uint path_idx, PathState *path_state) const {
    auto &isect = path_state->shape_intersections[path_idx].interaction;
    auto &lambda = path_state->lambdas[path_idx];

    auto &ray = path_state->camera_rays[path_idx].ray;
    auto sampler = &path_state->samplers[path_idx];

    if (regularize && path_state->mis_parameters[path_idx].any_non_specular_bounces) {
        path_state->bsdf[path_idx].regularize();
    }

    if (_is_non_specular(path_state->bsdf[path_idx].flags())) {
        SampledSpectrum Ld = sample_ld(isect, &path_state->bsdf[path_idx], lambda, sampler);
        path_state->L[path_idx] += path_state->beta[path_idx] * Ld;
    }

    // Sample BSDF to get new path direction
    Vector3f wo = -ray.d;
    FloatType u = sampler->get_1d();
    auto bs = path_state->bsdf[path_idx].sample_f(wo, u, sampler->get_2d());
    if (!bs) {
        path_state->beta[path_idx] = SampledSpectrum(0.0);
        return;
    }

    path_state->beta[path_idx] *= bs->f * bs->wi.abs_dot(isect.shading.n.to_vector3()) / bs->pdf;

    path_state->mis_parameters[path_idx].pdf_bsdf =
        bs->pdf_is_proportional ? path_state->bsdf[path_idx].pdf(wo, bs->wi) : bs->pdf;
    path_state->mis_parameters[path_idx].specular_bounce = bs->is_specular();
    path_state->mis_parameters[path_idx].any_non_specular_bounces |= (!bs->is_specular());

    if (bs->is_transmission()) {
        path_state->mis_parameters[path_idx].eta_scale *= sqr(bs->eta);
    }

    path_state->mis_parameters[path_idx].prev_interaction_light_sample_ctx = isect;

    path_state->camera_rays[path_idx].ray = isect.spawn_ray(bs->wi);
}

template <Material::Type material_type>
void WavefrontPathIntegrator::evaluate_material() {
    uint material_counter = 0;
    switch (material_type) {
    case Material::Type::coated_conductor: {
        material_counter = queues.coated_conductor_material_counter;
        break;
    }

    case Material::Type::coated_diffuse: {
        material_counter = queues.coated_diffuse_material_counter;
        break;
    }

    case Material::Type::conductor: {
        material_counter = queues.conductor_material_counter;
        break;
    }

    case Material::Type::dielectric: {
        material_counter = queues.dielectric_material_counter;
        break;
    }

    case Material::Type::diffuse: {
        material_counter = queues.diffuse_material_counter;
        break;
    }

    default: {
        REPORT_FATAL_ERROR();
    }
    }

    if (material_counter <= 0) {
        return;
    }

    const uint threads = 256;
    const auto blocks = divide_and_ceil(material_counter, threads);

    gpu_evaluate_material<material_type><<<blocks, threads>>>(this, &path_state, &queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

PBRT_CPU_GPU
void PathState::init_new_path(uint path_idx) {
    intersected[path_idx] = false;
    finished[path_idx] = false;

    L[path_idx] = SampledSpectrum(0.0);
    beta[path_idx] = SampledSpectrum(1.0);
    path_length[path_idx] = 0;

    mis_parameters[path_idx].init();
}

void PathState::create(uint samples_per_pixel, const Point2i &_resolution,
                       const std::string &sampler_type, std::vector<void *> &gpu_dynamic_pointers) {
    image_resolution = _resolution;
    global_path_counter = 0;
    total_path_num = samples_per_pixel * image_resolution.x * image_resolution.y;

    CHECK_CUDA_ERROR(hipMallocManaged(&camera_samples, sizeof(CameraSample) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&camera_rays, sizeof(CameraRay) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&lambdas, sizeof(SampledWavelengths) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(hipMallocManaged(&L, sizeof(SampledSpectrum) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&beta, sizeof(SampledSpectrum) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(
        hipMallocManaged(&shape_intersections, sizeof(ShapeIntersection) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(hipMallocManaged(&path_length, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&intersected, sizeof(bool) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&finished, sizeof(bool) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&pixel_indices, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&sample_indices, sizeof(uint) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(hipMallocManaged(&bsdf, sizeof(BSDF) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(hipMallocManaged(&mis_parameters, sizeof(MISParameter) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(hipMallocManaged(&samplers, sizeof(Sampler) * PATH_POOL_SIZE));

    for (auto ptr :
         std::vector<void *>({camera_samples, camera_rays, lambdas, L, beta, shape_intersections,
                              path_length, intersected, finished, pixel_indices, sample_indices,
                              bsdf, mis_parameters, samplers})) {
        gpu_dynamic_pointers.push_back(ptr);
    }

    const uint threads = 1024;
    uint blocks = divide_and_ceil<uint>(PATH_POOL_SIZE, threads);

    if (sampler_type == "stratified") {
        const uint samples_per_dimension = std::sqrt(samples_per_pixel);
        if (samples_per_dimension * samples_per_dimension != samples_per_pixel) {
            REPORT_FATAL_ERROR();
        }

        StratifiedSampler *stratified_samplers;
        CHECK_CUDA_ERROR(
            hipMallocManaged(&stratified_samplers, sizeof(StratifiedSampler) * PATH_POOL_SIZE));
        gpu_dynamic_pointers.push_back(stratified_samplers);

        gpu_init_stratified_samplers<<<blocks, threads>>>(samplers, stratified_samplers,
                                                          samples_per_dimension, PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else if (sampler_type == "independent") {
        IndependentSampler *independent_samplers;
        CHECK_CUDA_ERROR(
            hipMallocManaged(&independent_samplers, sizeof(IndependentSampler) * PATH_POOL_SIZE));
        gpu_dynamic_pointers.push_back(independent_samplers);

        gpu_init_independent_samplers<<<blocks, threads>>>(samplers, independent_samplers,
                                                           PATH_POOL_SIZE);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    } else {
        REPORT_FATAL_ERROR();
    }

    gpu_init_path_state<<<PATH_POOL_SIZE, threads>>>(this);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
}

void Queues::init(std::vector<void *> &gpu_dynamic_pointers) {
    CHECK_CUDA_ERROR(hipMallocManaged(&new_path_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&ray_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&frame_buffer_queue, sizeof(FrameBuffer) * PATH_POOL_SIZE));

    CHECK_CUDA_ERROR(
        hipMallocManaged(&coated_conductor_material_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(
        hipMallocManaged(&coated_diffuse_material_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&conductor_material_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&dielectric_material_queue, sizeof(uint) * PATH_POOL_SIZE));
    CHECK_CUDA_ERROR(hipMallocManaged(&diffuse_material_queue, sizeof(uint) * PATH_POOL_SIZE));

    for (auto ptr : std::vector<void *>({new_path_queue, ray_queue, frame_buffer_queue,
                                         coated_conductor_material_queue,
                                         coated_diffuse_material_queue, conductor_material_queue,
                                         dielectric_material_queue, diffuse_material_queue})) {
        gpu_dynamic_pointers.push_back(ptr);
    }
}

WavefrontPathIntegrator *
WavefrontPathIntegrator::create(const ParameterDictionary &parameters, const IntegratorBase *base,
                                const std::string &sampler_type, uint samples_per_pixel,
                                std::vector<void *> &gpu_dynamic_pointers) {
    WavefrontPathIntegrator *integrator;
    CHECK_CUDA_ERROR(hipMallocManaged(&integrator, sizeof(WavefrontPathIntegrator)));
    gpu_dynamic_pointers.push_back(integrator);

    if (sampler_type == "stratified") {
        auto old_spp = samples_per_pixel;
        samples_per_pixel = sqr(int(std::sqrt(samples_per_pixel)));
        if (old_spp != samples_per_pixel) {
            printf("samples per pixel adjusted: %d -> %d\n", old_spp, samples_per_pixel);
        }
    }

    integrator->samples_per_pixel = samples_per_pixel;

    integrator->base = base;
    integrator->path_state.create(samples_per_pixel, base->camera->get_camerabase()->resolution,
                                  sampler_type, gpu_dynamic_pointers);
    integrator->queues.init(gpu_dynamic_pointers);

    integrator->max_depth = parameters.get_integer("maxdepth", 5);

    integrator->regularize = parameters.get_bool("regularize", false);

    return integrator;
}

PBRT_GPU
SampledSpectrum WavefrontPathIntegrator::sample_ld(const SurfaceInteraction &intr, const BSDF *bsdf,
                                                   SampledWavelengths &lambda,
                                                   Sampler *sampler) const {
    // Initialize _LightSampleContext_ for light sampling
    LightSampleContext ctx(intr);
    // Try to nudge the light sampling position to correct side of the surface
    BxDFFlags flags = bsdf->flags();
    if (_is_reflective(flags) && !_is_transmissive(flags)) {
        ctx.pi = intr.offset_ray_origin(intr.wo);
    } else if (_is_transmissive(flags) && !_is_reflective(flags)) {
        ctx.pi = intr.offset_ray_origin(-intr.wo);
    }

    // Choose a light source for the direct lighting calculation
    FloatType u = sampler->get_1d();
    auto sampled_light = base->light_sampler->sample(ctx, u);

    Point2f uLight = sampler->get_2d();
    if (!sampled_light) {
        return SampledSpectrum(0);
    }

    // Sample a point on the light source for direct lighting
    auto light = sampled_light->light;
    auto ls = light->sample_li(ctx, uLight, lambda);
    if (!ls || !ls->l.is_positive() || ls->pdf == 0) {
        return SampledSpectrum(0);
    }

    // Evaluate BSDF for light sample and check light visibility
    Vector3f wo = intr.wo;
    Vector3f wi = ls->wi;
    SampledSpectrum f = bsdf->f(wo, wi) * wi.abs_dot(intr.shading.n.to_vector3());

    if (!f.is_positive() || !base->unoccluded(intr, ls->p_light)) {
        return SampledSpectrum(0);
    }

    // Return light's contribution to reflected radiance
    FloatType pdf_light = sampled_light->p * ls->pdf;
    if (pbrt::is_delta_light(light->get_light_type())) {
        return ls->l * f / pdf_light;
    }

    // for non delta light
    FloatType pdf_bsdf = bsdf->pdf(wo, wi);
    FloatType weight_light = power_heuristic(1, pdf_light, 1, pdf_bsdf);

    return weight_light * ls->l * f / pdf_light;
}

void WavefrontPathIntegrator::render(Film *film, const bool preview) {
    printf("wavefront: path pool size: %u\n", PATH_POOL_SIZE);

    const auto image_resolution = this->path_state.image_resolution;

    const auto num_pixels = image_resolution.x * image_resolution.y;

    std::vector<void *> gpu_dynamic_pointers;
    uint8_t *gpu_frame_buffer = nullptr;
    GLObject gl_object;
    if (preview) {
        gl_object.init("initializing", image_resolution);

        CHECK_CUDA_ERROR(hipMallocManaged(
            &gpu_frame_buffer, sizeof(uint8_t) * 3 * image_resolution.x * image_resolution.y));
        gpu_dynamic_pointers.push_back(gpu_frame_buffer);
    }

    constexpr uint threads = 256;

    // generate new paths for the whole pool
    fill_new_path_queue<<<PATH_POOL_SIZE, threads>>>(&queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    queues.new_path_counter = PATH_POOL_SIZE;

    queues.ray_counter = 0;
    generate_new_path<<<divide_and_ceil(queues.new_path_counter, threads), threads>>>(
        base, &path_state, &queues);
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    while (queues.ray_counter > 0) {
        ray_cast<<<divide_and_ceil(queues.ray_counter, threads), threads>>>(this, &path_state,
                                                                            &queues);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // clear all queues before control stage
        queues.new_path_counter = 0;
        queues.ray_counter = 0;
        queues.frame_buffer_counter = 0;

        queues.coated_conductor_material_counter = 0;
        queues.coated_diffuse_material_counter = 0;
        queues.conductor_material_counter = 0;
        queues.dielectric_material_counter = 0;
        queues.diffuse_material_counter = 0;

        control_logic<<<divide_and_ceil(PATH_POOL_SIZE, threads), threads>>>(this, &path_state,
                                                                             &queues);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        if (queues.frame_buffer_counter > 0) {
            // sort to make film writing deterministic
            std::sort(queues.frame_buffer_queue + 0,
                      queues.frame_buffer_queue + queues.frame_buffer_counter, FBComparator());

            write_frame_buffer<<<divide_and_ceil(queues.frame_buffer_counter, threads), threads>>>(
                film, &queues);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());

            if (preview) {
                film->copy_to_frame_buffer(gpu_frame_buffer);

                auto current_sample_idx =
                    std::min<uint>(path_state.global_path_counter / num_pixels, samples_per_pixel);

                gl_object.draw_frame(
                    gpu_frame_buffer,
                    GLObject::assemble_title(FloatType(current_sample_idx) / samples_per_pixel),
                    image_resolution);
            }
        }

        if (queues.new_path_counter > 0) {
            generate_new_path<<<divide_and_ceil(queues.new_path_counter, threads), threads>>>(
                base, &path_state, &queues);
            CHECK_CUDA_ERROR(hipDeviceSynchronize());
        }

        evaluate_material<Material::Type::coated_conductor>();

        evaluate_material<Material::Type::coated_diffuse>();

        evaluate_material<Material::Type::conductor>();

        evaluate_material<Material::Type::dielectric>();

        evaluate_material<Material::Type::diffuse>();
    }

    for (auto ptr : gpu_dynamic_pointers) {
        CHECK_CUDA_ERROR(hipFree(ptr));
    }
    CHECK_CUDA_ERROR(hipGetLastError());
}
