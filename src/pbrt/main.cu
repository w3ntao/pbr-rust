#include "hip/hip_runtime.h"
#include "pbrt/scene/builder.h"

using namespace std;

int main(int argc, const char **argv) {
    /*
    hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(RGBtoSpectrumData::RGBtoSpectrumTableGPU));
    hipDeviceSetLimit(hipLimitStackSize, 1024 * 8);

    size_t heapSize;
    hipDeviceGetLimit(&heapSize, hipLimitMallocHeapSize);

    size_t stackSize;
    hipDeviceGetLimit(&stackSize, hipLimitStackSize);

    printf("max stack size: %d\n", stackSize);
    printf("max heap size:  %d\n", heapSize);
    */

    const auto command_line_option = CommandLineOption(argc, argv);
    SceneBuilder::render_pbrt(command_line_option);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hipDeviceReset();

    return 0;
}
