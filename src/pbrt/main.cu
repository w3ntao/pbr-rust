#include "hip/hip_runtime.h"
#include "pbrt/scene/builder.h"

using namespace std;

int main(int argc, const char **argv) {
    {
        int runtime_version;
        hipRuntimeGetVersion(&runtime_version);

        int major = runtime_version / 1000;
        int minor = runtime_version % 1000 / 10;
        int patch = runtime_version % 10;

        /*
        hipDeviceSetLimit(hipLimitMallocHeapSize,
        sizeof(RGBtoSpectrumData::RGBtoSpectrumTableGPU)); hipDeviceSetLimit(hipLimitStackSize,
        1024 * 8);
        */

        size_t heap_size;
        hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);

        size_t stack_size;
        hipDeviceGetLimit(&stack_size, hipLimitStackSize);

        printf("CUDA info:\n");
        printf("    runtime version: %d.%d.%d\n", major, minor, patch);
        printf("    max stack size:  %zu\n", stack_size);
        printf("    max heap size:   %zu\n", heap_size);
        printf("\n");
        fflush(stdout);
    }

    const auto command_line_option = CommandLineOption(argc, argv);
    SceneBuilder::render_pbrt(command_line_option);

    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    hipDeviceReset();

    return 0;
}
