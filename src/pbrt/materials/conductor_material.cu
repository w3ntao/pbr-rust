#include "hip/hip_runtime.h"
#include "pbrt/materials/conductor_material.h"

#include "pbrt/base/float_texture.h"
#include "pbrt/base/material.h"
#include "pbrt/base/spectrum.h"

#include "pbrt/bxdfs/conductor_bxdf.h"

#include "pbrt/scene/parameter_dictionary.h"
#include "pbrt/spectrum_util/sampled_spectrum.h"

#include "pbrt/spectrum_util/global_spectra.h"

void ConductorMaterial::init(const ParameterDictionary &parameters,
                             std::vector<void *> &gpu_dynamic_pointers) {
    eta = parameters.get_spectrum_texture("eta", SpectrumType::Unbounded, gpu_dynamic_pointers);
    k = parameters.get_spectrum_texture("k", SpectrumType::Unbounded, gpu_dynamic_pointers);
    reflectance =
        parameters.get_spectrum_texture("reflectance", SpectrumType::Albedo, gpu_dynamic_pointers);

    if (reflectance && (eta || k)) {
        printf("ERROR: for ConductorMaterial, both `reflectance` and (`eta` and `k`) can't be "
               "provided\n");
        REPORT_FATAL_ERROR();
    }

    if (!reflectance) {
        if (!eta) {
            auto spectrum_cu_eta =
                parameters.get_spectrum("metal-Cu-eta", SpectrumType::Albedo, gpu_dynamic_pointers);
            eta = SpectrumTexture::create_constant_texture(spectrum_cu_eta, gpu_dynamic_pointers);
        }

        if (!k) {
            auto spectrum_cu_k =
                parameters.get_spectrum("metal-Cu-k", SpectrumType::Albedo, gpu_dynamic_pointers);
            k = SpectrumTexture::create_constant_texture(spectrum_cu_k, gpu_dynamic_pointers);
        }
    }

    u_roughness = parameters.get_float_texture("uroughness", gpu_dynamic_pointers);
    if (!u_roughness) {
        auto roughness_val = parameters.get_float("roughness", 0.0);
        u_roughness =
            FloatTexture::create_constant_float_texture(roughness_val, gpu_dynamic_pointers);
    }

    v_roughness = parameters.get_float_texture("vroughness", gpu_dynamic_pointers);
    if (!v_roughness) {
        auto roughness_val = parameters.get_float("roughness", 0.0);
        v_roughness =
            FloatTexture::create_constant_float_texture(roughness_val, gpu_dynamic_pointers);
    }

    remap_roughness = parameters.get_bool("remaproughness", true);

    if (!u_roughness || !v_roughness) {
        REPORT_FATAL_ERROR();
    }
}

PBRT_GPU
ConductorBxDF ConductorMaterial::get_conductor_bsdf(const MaterialEvalContext &ctx,
                                                    SampledWavelengths &lambda) {
    auto uRough = u_roughness->evaluate(ctx);
    auto vRough = v_roughness->evaluate(ctx);

    if (remap_roughness) {
        uRough = TrowbridgeReitzDistribution::RoughnessToAlpha(uRough);
        vRough = TrowbridgeReitzDistribution::RoughnessToAlpha(vRough);
    }

    SampledSpectrum etas, ks;
    if (eta) {
        etas = eta->evaluate(ctx, lambda);
        ks = k->evaluate(ctx, lambda);

    } else {
        // Avoid r==0 NaN case...
        auto r = reflectance->evaluate(ctx, lambda).clamp(0, 0.9999);
        etas = SampledSpectrum(1.f);
        ks = 2 * r.sqrt() / (SampledSpectrum(1) - r).clamp(0, Infinity).sqrt();
    }
    TrowbridgeReitzDistribution distrib(uRough, vRough);
    return ConductorBxDF(distrib, etas, ks);
}
