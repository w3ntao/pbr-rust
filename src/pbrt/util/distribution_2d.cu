#include "hip/hip_runtime.h"
#include "pbrt/util/distribution_2d.h"

#include "pbrt/spectrum_util/rgb.h"
#include "pbrt/util/distribution_1d.h"
#include "pbrt/textures/gpu_image.h"

const Distribution2D *Distribution2D::create_from_image(const GPUImage *image,
                                                        std::vector<void *> &gpu_dynamic_pointers) {
    Distribution2D *distribution;
    CHECK_CUDA_ERROR(hipMallocManaged(&distribution, sizeof(Distribution2D)));
    gpu_dynamic_pointers.push_back(distribution);

    distribution->build_from_image(image, gpu_dynamic_pointers);

    return distribution;
}

void Distribution2D::build_from_image(const GPUImage *image,
                                      std::vector<void *> &gpu_dynamic_pointers) {
    dimension = image->get_resolution();

    distribution_1d_list = nullptr;
    cdf = nullptr;
    pmf = nullptr;

    FloatType max_luminance = 0.0;
    std::vector<std::vector<FloatType>> image_luminance_array(dimension.x,
                                                              std::vector<FloatType>(dimension.y));
    for (int x = 0; x < dimension.x; ++x) {
        for (int y = 0; y < dimension.y; ++y) {
            const auto rgb =
                image->fetch_pixel(Point2i(x, y), WrapMode::OctahedralSphere).clamp(0, Infinity);

            auto luminance = rgb.avg();
            image_luminance_array[x][y] = luminance;

            max_luminance = std::max(max_luminance, luminance);
        }
    }

    if (max_luminance <= 0.0) {
        REPORT_FATAL_ERROR();
    }

    FloatType *_pmf;
    CHECK_CUDA_ERROR(hipMallocManaged(&_pmf, sizeof(FloatType) * dimension.x));
    gpu_dynamic_pointers.push_back(_pmf);

    // ignore minimal values
    // those pixels with luminance smaller than 0.01 * max_luminance are ignored
    const auto ignore_ratio = 0.01;
    double sum_pmf = 0.0;
    auto num_ignore = 0;
    auto ignore_threshold = ignore_ratio * max_luminance;
    for (int x = 0; x < dimension.x; ++x) {
        FloatType luminance_per_row = 0.0;
        for (int y = 0; y < dimension.y; ++y) {
            if (image_luminance_array[x][y] <= ignore_threshold) {
                image_luminance_array[x][y] = 0.0;
                num_ignore += 1;
                continue;
            }

            luminance_per_row += image_luminance_array[x][y];
        }

        _pmf[x] = luminance_per_row;
        sum_pmf += luminance_per_row;
    }

    auto num_pixels = dimension.x * dimension.y;
    printf("%s(): %d/%d (%.2f%) values ignored (ignored ratio: %f)\n", __func__, num_ignore,
           num_pixels, FloatType(num_ignore) / num_pixels * 100, ignore_ratio);

    for (uint idx = 0; idx < dimension.x; ++idx) {
        _pmf[idx] = _pmf[idx] / sum_pmf;
    }

    FloatType *_cdf;
    CHECK_CUDA_ERROR(hipMallocManaged(&_cdf, sizeof(FloatType) * dimension.x));
    gpu_dynamic_pointers.push_back(_cdf);

    _cdf[0] = _pmf[0];
    for (uint idx = 1; idx < dimension.x; ++idx) {
        _cdf[idx] = _cdf[idx - 1] + _pmf[idx];
    }

    pmf = _pmf;
    cdf = _cdf;

    Distribution1D *_distribution_1d_list;
    CHECK_CUDA_ERROR(
        hipMallocManaged(&_distribution_1d_list, sizeof(Distribution1D) * dimension.x));
    gpu_dynamic_pointers.push_back(_distribution_1d_list);

    for (int x = 0; x < dimension.x; ++x) {
        std::vector<FloatType> pdfs(dimension.y);
        for (int y = 0; y < dimension.y; ++y) {
            const auto rgb =
                image->fetch_pixel(Point2i(x, y), WrapMode::OctahedralSphere).clamp(0, Infinity);
            pdfs[y] = rgb.avg();
        }

        _distribution_1d_list[x].build(pdfs, gpu_dynamic_pointers);
    }

    distribution_1d_list = _distribution_1d_list;
}

PBRT_GPU
cuda::std::pair<Point2f, FloatType> Distribution2D::sample(const Point2f &uv) const {
    auto first_dim_idx = search_cdf(uv.x, cdf, dimension.x);

    auto first_pdf = pmf[first_dim_idx];

    auto second_dim_result = distribution_1d_list[first_dim_idx].sample(uv.y);

    auto pdf = first_pdf * second_dim_result.second;

    return {Point2f(FloatType(first_dim_idx) / FloatType(dimension.x),
                    FloatType(second_dim_result.first) / FloatType(dimension.y)),
            pdf};
}

PBRT_CPU_GPU
FloatType Distribution2D::get_pdf(const Point2f &u) const {
    auto first_dim_index = clamp<uint>(u.x * FloatType(dimension.x), 0, dimension.x - 1);
    auto first_dim_pdf = pmf[first_dim_index];

    auto second_dim_index = clamp<uint>(u.y * FloatType(dimension.y), 0, dimension.y - 1);
    auto second_dim_pdf = distribution_1d_list[first_dim_index].get_pdf(second_dim_index);

    return first_dim_pdf * second_dim_pdf;
}
