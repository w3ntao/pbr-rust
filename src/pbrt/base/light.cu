#include "hip/hip_runtime.h"
#include "pbrt/base/light.h"
#include "pbrt/base/shape.h"
#include "pbrt/lights/diffuse_area_light.h"
#include "pbrt/lights/distant_light.h"
#include "pbrt/lights/image_infinite_light.h"

template <typename TypeOfLight>
static __global__ void init_lights(Light *lights, TypeOfLight *concrete_lights, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    lights[worker_idx].init(&concrete_lights[worker_idx]);
}

Light *Light::create(const std::string &type_of_light, const Transform &render_from_light,
                     const ParameterDictionary &parameters,
                     std::vector<void *> &gpu_dynamic_pointers) {
    if (type_of_light == "distant") {
        auto distant_light =
            DistantLight::create(render_from_light, parameters, gpu_dynamic_pointers);

        Light *light;
        CHECK_CUDA_ERROR(hipMallocManaged(&light, sizeof(Light)));
        gpu_dynamic_pointers.push_back(light);

        light->init(distant_light);
        return light;
    }

    if (type_of_light == "infinite") {
        auto image_infinite_light =
            ImageInfiniteLight::create(render_from_light, parameters, gpu_dynamic_pointers);

        Light *light;
        CHECK_CUDA_ERROR(hipMallocManaged(&light, sizeof(Light)));
        gpu_dynamic_pointers.push_back(light);

        light->init(image_infinite_light);
        return light;
    }

    printf("\n%s(): Light `%s` not implemented\n", __func__, type_of_light.c_str());
    REPORT_FATAL_ERROR();
    return nullptr;
}

Light *Light::create_diffuse_area_lights(const Shape *shapes, const uint num,
                                         const Transform &render_from_light,
                                         const ParameterDictionary &parameters,
                                         std::vector<void *> &gpu_dynamic_pointers) {
    const uint threads = 1024;
    const uint blocks = divide_and_ceil(num, threads);

    // build DiffuseAreaLight
    DiffuseAreaLight *diffuse_area_lights;
    CHECK_CUDA_ERROR(hipMallocManaged(&diffuse_area_lights, sizeof(DiffuseAreaLight) * num));
    Light *lights;
    CHECK_CUDA_ERROR(hipMallocManaged(&lights, sizeof(Light) * num));

    gpu_dynamic_pointers.push_back(diffuse_area_lights);
    gpu_dynamic_pointers.push_back(lights);

    for (uint idx = 0; idx < num; idx++) {
        diffuse_area_lights[idx].init(&shapes[idx], render_from_light, parameters,
                                      gpu_dynamic_pointers);
    }

    init_lights<<<blocks, threads>>>(lights, diffuse_area_lights, num);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    return lights;
}

PBRT_CPU_GPU
void Light::init(DistantLight *distant_light) {
    type = Type::distant_light;
    ptr = distant_light;
}

PBRT_CPU_GPU
void Light::init(DiffuseAreaLight *diffuse_area_light) {
    type = Type::diffuse_area_light;
    ptr = diffuse_area_light;
}

PBRT_CPU_GPU
void Light::init(ImageInfiniteLight *image_infinite_light) {
    type = Type::image_infinite_light;
    ptr = image_infinite_light;
}

PBRT_CPU_GPU
LightType Light::get_light_type() const {
    switch (type) {
    case (Type::diffuse_area_light): {
        return ((DiffuseAreaLight *)ptr)->get_light_type();
    }

    case (Type::distant_light): {
        return ((DistantLight *)ptr)->get_light_type();
    }

    case (Type::image_infinite_light): {
        return ((ImageInfiniteLight *)ptr)->get_light_type();
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_GPU
SampledSpectrum Light::l(Point3f p, Normal3f n, Point2f uv, Vector3f w,
                         const SampledWavelengths &lambda) const {
    switch (type) {
    case (Type::diffuse_area_light): {
        return ((DiffuseAreaLight *)ptr)->l(p, n, uv, w, lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_GPU
SampledSpectrum Light::le(const Ray &ray, const SampledWavelengths &lambda) const {
    switch (type) {
    case (Type::image_infinite_light): {
        return ((ImageInfiniteLight *)ptr)->le(ray, lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_GPU
cuda::std::optional<LightLiSample> Light::sample_li(const LightSampleContext &ctx, const Point2f &u,
                                                    SampledWavelengths &lambda) const {
    switch (type) {
    case (Type::diffuse_area_light): {
        return ((DiffuseAreaLight *)ptr)->sample_li(ctx, u, lambda);
    }

    case (Type::distant_light): {
        return ((DistantLight *)ptr)->sample_li(ctx, u, lambda);
    }

    case (Type::image_infinite_light): {
        return ((ImageInfiniteLight *)ptr)->sample_li(ctx, u, lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_GPU
FloatType Light::pdf_li(const LightSampleContext &ctx, const Vector3f &wi,
                        bool allow_incomplete_pdf) const {
    switch (type) {
    case (Type::diffuse_area_light): {
        return ((DiffuseAreaLight *)ptr)->pdf_li(ctx, wi, allow_incomplete_pdf);
    }

    case (Type::image_infinite_light): {
        return ((ImageInfiniteLight *)ptr)->pdf_li(ctx, wi, allow_incomplete_pdf);
    }
    }
    REPORT_FATAL_ERROR();
    return NAN;
}

PBRT_CPU_GPU
SampledSpectrum Light::phi(const SampledWavelengths &lambda) const {
    switch (type) {
    case (Type::diffuse_area_light): {
        return ((DiffuseAreaLight *)ptr)->phi(lambda);
    }

    case (Type::distant_light): {
        return ((DistantLight *)ptr)->phi(lambda);
    }

    case (Type::image_infinite_light): {
        return ((ImageInfiniteLight *)ptr)->phi(lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

void Light::preprocess(const Bounds3<FloatType> &scene_bounds) {
    switch (type) {
    case (Type::diffuse_area_light): {
        // do nothing
        return;
    }

    case (Type::distant_light): {
        ((DistantLight *)ptr)->preprocess(scene_bounds);
        return;
    }

    case (Type::image_infinite_light): {
        ((ImageInfiniteLight *)ptr)->preprocess(scene_bounds);
        return;
    }
    }

    REPORT_FATAL_ERROR();
}
