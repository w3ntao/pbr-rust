#include "pbrt/base/primitive.h"

#include "pbrt/primitives/simple_primitives.h"
#include "pbrt/primitives/geometric_primitive.h"

const Primitive *Primitive::create_simple_primitive(const Shape *shape, const Material *material,
                                                    std::vector<void *> &gpu_dynamic_pointers) {
    SimplePrimitive *simple_primitive;
    CHECK_CUDA_ERROR(hipMallocManaged(&simple_primitive, sizeof(SimplePrimitive)));
    Primitive *primitive;
    CHECK_CUDA_ERROR(hipMallocManaged(&primitive, sizeof(Primitive)));

    gpu_dynamic_pointers.push_back(simple_primitive);
    gpu_dynamic_pointers.push_back(primitive);

    simple_primitive->init(shape, material);
    primitive->init(simple_primitive);

    return primitive;
}

PBRT_CPU_GPU
void Primitive::init(const SimplePrimitive *simple_primitive) {
    type = Type::simple_primitive;
    ptr = simple_primitive;
}

PBRT_CPU_GPU
void Primitive::init(const GeometricPrimitive *geometric_primitive) {
    type = Type::geometric_primitive;
    ptr = geometric_primitive;
}

PBRT_CPU_GPU
Bounds3f Primitive::bounds() const {
    switch (type) {
    case (Type::simple_primitive): {
        return ((SimplePrimitive *)ptr)->bounds();
    }

    case (Type::geometric_primitive): {
        return ((GeometricPrimitive *)ptr)->bounds();
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}

PBRT_GPU
bool Primitive::fast_intersect(const Ray &ray, FloatType t_max) const {
    switch (type) {
    case (Type::simple_primitive): {
        return ((SimplePrimitive *)ptr)->fast_intersect(ray, t_max);
    }

    case (Type::geometric_primitive): {
        return ((GeometricPrimitive *)ptr)->fast_intersect(ray, t_max);
    }
    }

    REPORT_FATAL_ERROR();
    return false;
}

PBRT_GPU
cuda::std::optional<ShapeIntersection> Primitive::intersect(const Ray &ray, FloatType t_max) const {
    switch (type) {
    case (Type::simple_primitive): {
        return ((SimplePrimitive *)ptr)->intersect(ray, t_max);
    }

    case (Type::geometric_primitive): {
        return ((GeometricPrimitive *)ptr)->intersect(ray, t_max);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}
