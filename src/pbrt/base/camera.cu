#include "pbrt/base/camera.h"
#include "pbrt/cameras/perspective.h"

Camera *Camera::create_perspective_camera(const Point2i &resolution,
                                          const CameraTransform &camera_transform, FloatType fov,
                                          FloatType lens_radius,
                                          std::vector<void *> &gpu_dynamic_pointers) {
    PerspectiveCamera *perspective_camera;
    CHECK_CUDA_ERROR(hipMallocManaged(&perspective_camera, sizeof(PerspectiveCamera)));

    Camera *camera;
    CHECK_CUDA_ERROR(hipMallocManaged(&camera, sizeof(Camera)));

    gpu_dynamic_pointers.push_back(perspective_camera);
    gpu_dynamic_pointers.push_back(camera);

    perspective_camera->init(resolution, camera_transform, fov, lens_radius);
    camera->init(perspective_camera);

    return camera;
}

void Camera::init(const PerspectiveCamera *perspective_camera) {
    ptr = perspective_camera;
    type = Type::perspective;
}

PBRT_CPU_GPU
const CameraBase *Camera::get_camerabase() const {
    switch (type) {
    case (Type::perspective): {
        return &((PerspectiveCamera *)ptr)->camera_base;
    }
    }

    REPORT_FATAL_ERROR();
    return nullptr;
}

PBRT_CPU_GPU
CameraRay Camera::generate_ray(const CameraSample &sample) const {
    switch (type) {
    case (Type::perspective): {
        return ((PerspectiveCamera *)ptr)->generate_ray(sample);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}
