#include "pbrt/base/megakernel_integrator.h"
#include "pbrt/base/ray.h"
#include "pbrt/base/sampler.h"
#include "pbrt/integrators/ambient_occlusion.h"
#include "pbrt/integrators/path.h"
#include "pbrt/integrators/random_walk.h"
#include "pbrt/integrators/simple_path.h"
#include "pbrt/integrators/surface_normal.h"
#include "pbrt/spectrum_util/sampled_wavelengths.h"

const Integrator *Integrator::create(const ParameterDictionary &parameters,
                                     const std::string &integrator_name,
                                     const IntegratorBase *integrator_base,
                                     std::vector<void *> &gpu_dynamic_pointers) {

    Integrator *integrator;
    CHECK_CUDA_ERROR(hipMallocManaged(&integrator, sizeof(Integrator)));
    gpu_dynamic_pointers.push_back(integrator);

    if (integrator_name == "ambientocclusion") {
        auto ambient_occlusion_integrator =
            AmbientOcclusionIntegrator::create(parameters, integrator_base, gpu_dynamic_pointers);

        integrator->init(ambient_occlusion_integrator);
        return integrator;
    }

    if (integrator_name == "path") {
        auto path_integrator =
            PathIntegrator::create(parameters, integrator_base, gpu_dynamic_pointers);

        integrator->init(path_integrator);
        return integrator;
    }

    if (integrator_name == "surfacenormal") {
        auto surface_normal_integrator =
            SurfaceNormalIntegrator::create(parameters, integrator_base, gpu_dynamic_pointers);

        integrator->init(surface_normal_integrator);
        return integrator;
    }

    if (integrator_name == "simplepath") {
        auto simple_path_integrator =
            SimplePathIntegrator::create(parameters, integrator_base, gpu_dynamic_pointers);

        integrator->init(simple_path_integrator);
        return integrator;
    }

    printf("\n%s(): unknown Integrator: %s\n\n", __func__, integrator_name.c_str());
    REPORT_FATAL_ERROR();
    return nullptr;
}

void Integrator::init(const AmbientOcclusionIntegrator *ambient_occlusion_integrator) {
    type = Type::ambient_occlusion;
    ptr = ambient_occlusion_integrator;
}

void Integrator::init(const PathIntegrator *path_integrator) {
    type = Type::path;
    ptr = path_integrator;
}

void Integrator::init(const RandomWalkIntegrator *random_walk_integrator) {
    type = Type::random_walk;
    ptr = random_walk_integrator;
}

void Integrator::init(const SurfaceNormalIntegrator *surface_normal_integrator) {
    type = Type::surface_normal;
    ptr = surface_normal_integrator;
}

void Integrator::init(const SimplePathIntegrator *simple_path_integrator) {
    type = Type::simple_path;
    ptr = simple_path_integrator;
}

PBRT_GPU
SampledSpectrum Integrator::li(const Ray &ray, SampledWavelengths &lambda, Sampler *sampler) const {
    switch (type) {
    case (Type::ambient_occlusion): {
        return ((AmbientOcclusionIntegrator *)ptr)->li(ray, lambda, sampler);
    }

    case (Type::path): {
        return ((PathIntegrator *)ptr)->li(ray, lambda, sampler);
    }

    case (Type::random_walk): {
        return ((RandomWalkIntegrator *)ptr)->li(ray, lambda, sampler);
    }

    case (Type::simple_path): {
        return ((SimplePathIntegrator *)ptr)->li(ray, lambda, sampler);
    }

    case (Type::surface_normal): {
        return ((SurfaceNormalIntegrator *)ptr)->li(ray, lambda);
    }
    }

    REPORT_FATAL_ERROR();
    return {};
}
