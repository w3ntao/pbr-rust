#include "hip/hip_runtime.h"
#include "pbrt/shapes/triangle_mesh.h"

#include "pbrt/base/shape.h"
#include "pbrt/shapes/triangle.h"

template <typename T>
static __global__ void apply_transform(T *data, const Transform transform, uint length) {
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= length) {
        return;
    }

    data[idx] = transform(data[idx]);
}

static __global__ void init_triangles_from_mesh(Triangle *triangles, const TriangleMesh *mesh) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= mesh->triangles_num) {
        return;
    }

    triangles[worker_idx].init(worker_idx, mesh);
}

template <typename TypeOfShape>
static __global__ void init_shapes(Shape *shapes, const TypeOfShape *concrete_shapes, uint num) {
    const uint worker_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (worker_idx >= num) {
        return;
    }

    shapes[worker_idx].init(&concrete_shapes[worker_idx]);
}

std::pair<const Shape *, uint>
TriangleMesh::build_triangles(const Transform &render_from_object, bool reverse_orientation,
                              const std::vector<Point3f> &points, const std::vector<int> &indices,
                              const std::vector<Point2f> &uv,
                              std::vector<void *> &gpu_dynamic_pointers) {
    Point3f *gpu_points;
    CHECK_CUDA_ERROR(hipMallocManaged(&gpu_points, sizeof(Point3f) * points.size()));
    CHECK_CUDA_ERROR(hipMemcpy(gpu_points, points.data(), sizeof(Point3f) * points.size(),
                                hipMemcpyHostToDevice));
    const uint threads = 1024;
    {
        const uint blocks = divide_and_ceil<uint>(points.size(), threads);
        apply_transform<<<blocks, threads>>>(gpu_points, render_from_object, points.size());
    }
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    int *gpu_indices;
    CHECK_CUDA_ERROR(hipMallocManaged(&gpu_indices, sizeof(int) * indices.size()));
    CHECK_CUDA_ERROR(hipMemcpy(gpu_indices, indices.data(), sizeof(int) * indices.size(),
                                hipMemcpyHostToDevice));

    Point2f *gpu_uv = nullptr;
    if (!uv.empty()) {
        CHECK_CUDA_ERROR(hipMallocManaged(&gpu_uv, sizeof(Point2f) * uv.size()));
        CHECK_CUDA_ERROR(
            hipMemcpy(gpu_uv, uv.data(), sizeof(Point2f) * uv.size(), hipMemcpyHostToDevice));
        gpu_dynamic_pointers.push_back(gpu_uv);
    }

    TriangleMesh *mesh;
    CHECK_CUDA_ERROR(hipMallocManaged(&mesh, sizeof(TriangleMesh)));
    mesh->init(reverse_orientation, gpu_indices, indices.size(), gpu_points, gpu_uv);

    uint num_triangles = mesh->triangles_num;
    Triangle *triangles;
    CHECK_CUDA_ERROR(hipMallocManaged(&triangles, sizeof(Triangle) * num_triangles));
    Shape *shapes;
    CHECK_CUDA_ERROR(hipMallocManaged(&shapes, sizeof(Shape) * num_triangles));

    {
        const uint blocks = divide_and_ceil(num_triangles, threads);
        init_triangles_from_mesh<<<blocks, threads>>>(triangles, mesh);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        init_shapes<<<blocks, threads>>>(shapes, triangles, num_triangles);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    }

    for (auto ptr : std::vector<void *>({
             gpu_indices,
             gpu_points,
             mesh,
             triangles,
             shapes,
         })) {
        gpu_dynamic_pointers.push_back(ptr);
    }

    return {shapes, num_triangles};
}